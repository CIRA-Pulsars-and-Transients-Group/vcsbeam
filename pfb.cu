#include "hip/hip_runtime.h"
/********************************************************
 *                                                      *
 * Licensed under the Academic Free License version 3.0 *
 *                                                      *
 ********************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>

extern "C" {
#include "pfb.h"
#include "filter.h"
#include "jones.h"
}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    /* Wrapper function for GPU/CUDA error handling. Every CUDA call goes through
       this function. It will return a message giving your the error string,
       file name and line of the error. Aborts on error. */

    if (code != 0)
    {
        fprintf(stderr, "GPUAssert:: %s - %s (%d)\n", hipGetErrorString(code), file, line);
        if (abort)
        {
            exit(code);
        }
    }
}

// define a macro for accessing gpuAssert
#define gpuErrchk(ans) {gpuAssert((ans), __FILE__, __LINE__, true);}

/*******************************
 * FORWARD (ANALYSIS) FINE PFB *
 *******************************/

/* The following kernels are part of an "offline" version of the "fine PFB"
   algorithm that was implemented on the FPGAs of Phase 1 & 2 of the MWA. As
   described in McSweeney et al. (2020), this algorithm is a version of the
   "weighted overlap-add" algorithm (see their Eq. (3)):

              K-1
     X_k[m] = SUM b_m[n] e^(-2πjkn/K),
              n=0

   where

              P-1
     b_m[n] = SUM h[Kρ-n] x[n+mM-Kρ].
              ρ=0

   A full description of these symbols is given in the reference, but it
   should be noted here that, for the kernels below,

     - "x" represents the input data (INDATA),
     - "X" represents the output data (OUTDATA),
     - "h" represents the filter coefficients (FILTER_COEFFS),
     - "P" represents the number of taps (NTAPS)
     - "b" represents the weighted overlap-add array (WEIGHTED_OVERLAP_ARRAY)

   The algorithm is broken up into three parts:
     (1) the weighted overlap-add, which forms "b" from "x" and "h",
     (2) the FFT, which is implemented using the cuFFT library, and
     (3) the demotion and packaging of the result into the required
         output format, which is described in the Appendix of McSweeney et al.
         (2020).

   Notes:

    - INDATA is expected to have the same data layout as delivered by mwalib's
      mwalib_voltage_context_read_second() function when applied to MWAX high
      time resolution data, whose format description can be found at
      https://wiki.mwatelescope.org/display/MP/MWA+High+Time+Resolution+Voltage+Capture+System
      The samples are organised according to the vMWAX_IDX macro.

    - OUTDATA will have the same data layout as "recombined" legacy VCS data:
      (4+4)-bit complex samples, with imaginary component occupying the first
      4 bits, and samples organised according to the v_IDX macro.

    - Each thread will operate on one RF input (i.e. antenna/pol combination)
      and generate the spectrum for a single "fine-channelised" time step
      (the "m" index). The "weighted overlap-add" array ("b") also resides in
      device memory, in order that it can make use of cuFFT. This is a 
 */

__global__ void legacy_pfb_weighted_overlap_add( char2 *indata,
        int *filter_coeffs, hipFloatComplex *weighted_overlap_array )
{
    // Parse the block and thread idxs:
    //   <<<(nspectra,I),(K,P)>>>
    // where nspectra is the number of output spectra,
    //       I        is the number of RF inputs,
    //       K        is the size of the output spectrum
    //       P        is the number of taps
    // ...and put everything into the mathematical notation used in McSweeney
    // et al. (2020) (see equation in comments above)
    int              m        = blockIdx.x;
    int              nspectra = gridDim.x;
    int              I        = gridDim.y;
    int              i        = blockIdx.y;
    int              K        = blockDim.x;
    int              M        = K; // This enforces a critical sampled PFB
    //int              P        = blockDim.y;
    int              n        = threadIdx.x;
    int              p        = threadIdx.y;

    int             *h = filter_coeffs;
    char2           *x = indata;
    hipFloatComplex  *b = weighted_overlap_array;

    // Use shared memory as a temporary workspace for preparing the b array
    // For one block, this should have K elements
    extern __shared__ int2 bint[];

    // Let the first tap (p=0) have the responsibility of initialising the
    // bint array to zeros
    if (p == 0)
    {
        bint[n].x = 0;
        bint[n].y = 0;
    }
    __syncthreads();

    // Now calculate the index into the various arrays that also
    // takes into account the fact that these arrays contain all RF inputs.
    // MEMO TO SELF: My current going theory is that I don't have to do any
    // re-ordering of the antennas, as that is dealt with elsewhere.
    int h_idx = K*p - n;
    int x_idx = vMWAX_IDX(n + m*M - K*p, i, I);
    int b_idx = (K*i + m)*nspectra + n; // This puts each set of K samples to
                                        // be FFT'd in a contiguous memory block

    // Now perform the weighted overlap-add operation
    int   hval = h[h_idx];
    char2 xval = x[x_idx];

    atomicAdd( &bint[n].x, hval*(int)xval.x );
    atomicAdd( &bint[n].y, hval*(int)xval.y );

    __syncthreads();

    // In keeping with the original FPGA implementation, the result now needs to
    // be demoted and rounded. Only one tap needs to do this
    int X, Y; // To avoid too many shared memory accesses
    if (p == 0)
    {
        X = bint[n].x;
        Y = bint[n].y;

        // Rounding:
        if (X > 0)  X += 0x2000;
        if (Y > 0)  Y += 0x2000;

        // Demotion:
        X >>= 14;
        Y >>= 14;

        // Promote the result to doubles and put it in the b array in global memory
        // in preparation for being FFTed
        b[b_idx] = make_hipFloatComplex( (double)X, (double)Y );
    }

    __syncthreads();
}

__global__ void pack_into_recombined_format( hipFloatComplex *ffted, uint8_t *outdata )
/* This is the final step in the forward fine PFB algorithm that emulates what
   was implemented on the MWA FPGAs in Phase 1 & 2 (see above for details).
   At this point, the FFTED array contains the Fourier-transformed data that
   already represents the final channelisation. All that remains to be done is
   to pack it into the same format as the VCS recombined data.

   Kernel signature:
     <<<(nspectra,K),I>>>
   where
     nspectra is the number of (fine-channelised) time samples
     K        is the number of channels
     I        is the number of RF inputs
*/
{
    // Parse the kernel signature, using the same mathematical notation
    // described above
    int nspectra = gridDim.x;
    int K        = gridDim.y;
    int m        = blockIdx.x;
    int k        = blockIdx.y;
    int i        = threadIdx.x;
    int I        = blockIdx.x;

    hipFloatComplex  *b = ffted;
    uint8_t         *X = outdata;

    // Calculate the idxs into b and X
    int b_idx = (K*i + m)*nspectra + k;
    int X_idx = v_IDX(m, k, i, K, I);

    // Pull the values to be manipulated into register memory (because the
    // packing macro below involves a lot of repetition of the arguments)
    double re = b[b_idx].x;
    double im = b[b_idx].y;

    // Put the packed value back into global memory at the appropriate place
    X[X_idx] = PACK_NIBBLES(re, im);

    __syncthreads();
}

forward_pfb *init_forward_pfb( MetafitsMetadata *obs_metadata,
        char2 *htr_data, char2 *htr_data_extended, uint8_t *vcs_data,
        pfb_filter *filter )
/* Create and initialise a forward_pfb struct.

   Inputs:
     OBS_METADATA      - mwalib metadata struct
     HTR_DATA          - pointer to host memory to be PFB'd
     HTR_DATA_EXTENDED - pointer to extended host memory to be PFB'd
                         (will be tacked onto the end of

   Output:
     FPFB         - Pointer to struct to be initialised
 */
{
    // Create the struct in memory
    forward_pfb *fpfb = (forward_pfb *)malloc( sizeof(forward_pfb) );

    // Host memory is assumed to be allocated
    fpfb->htr_data          = htr_data;
    fpfb->htr_data_extended = htr_data_extended;
    fpfb->vcs_data          = vcs_data;
}

void free_forward_pfb( forward_pfb *pfb )
// Free the memory allocated in init_forward_pfb
{
    free( fpfb );
}

void cu_forward_pfb_fpga_version( forward_pfb *fpfb, bool copy_result_to_host )
/* The wrapper function that performs the forward PFB algorithm as originally
   implemented on the FPGAs for MWA Phases 1 & 2.
   A cuFFT plan must already have been made, via make_forward_pfb_fpga_fft_plan().
 */
{
    // Copy data to device
    gpuErrchk(hipMemcpy( fpfb->d_htr_data, fpfb->htr_data, fpfb->htr_size, hipMemcpyHostToDevice ));
    if (fpfb->htr_data_extended != NULL)
        gpuErrchk(hipMemcpy( fpfb->d_htr_data + fpfb->htr_size, fpfb->htr_data_extended, fpfb->htr_extended_size, hipMemcpyHostToDevice ));

    // PFB algorithm:
    // 1st step: weighted overlap add
    dim3 blocks( fpfb->nspectra, fpfb->I );
    dim3 threads( fpfb->K, fpfb->P );

    legacy_pfb_weighted_overlap_add<<<blocks, threads>>>( fpfb->d_htr_data, fpfb->d_filter_coeffs, fpfb->d_weighted_overlap_add );
    gpuErrchk( hipPeekAtLastError() );

    // 2nd step: FFT
    hipfftExecC2C( fpfb->plan, fpfb->d_weighted_overlap_add, fpfb->d_weighted_overlap_add, HIPFFT_FORWARD );

    // 3rd step: packaging the result
    dim3 blocks2( fpfb->nspectra, fpfb->K );
    dim3 threads2( fpfb->I );

    pack_into_recombined_format<<<blocks2, threads2>>>( fpfb->d_weighted_overlap_add, fpfb->d_vcs_data );
    gpuErrchk( hipPeekAtLastError() );

    // Finally, copy the answer back to host memory, if requested
    if (copy_result_to_host)
    {
        gpuErrchk(hipMemcpy( fpfb->vcs_data, fpfb->d_vcs_data, fpfb->vcs_size, hipMemcpyDeviceToHost ));
    }
}

/**********************************
 * BACKWARDS (SYNTHESIS) FINE PFB *
 **********************************/

__global__ void ipfb_kernel(
    float *in_real, float *in_imag,
    float *ft_real, float *ft_imag,
    int ntaps, int npol, float *out )
/* This kernel computes the synthesis filter:

              1              K-1
   xhat[n] = --- SUM f[n-mM] SUM X_k[m] e^(2πjkn/K)
              K   m          k=0

   The sum over m is nominally over all integers, but in practice only
   involves a few terms because of the finiteness of the filter, f. To be
   precise, there are precisely ntaps non-zero values.

   X_k[m] represents the complex-valued inputs, in_real and in_imag.
   Every possible value of f[n]*e^(2πjkn/K) is provided in ft_real and
   ft_imag.

   K is the number of channels, and because this is a critically sampled
   PFB, M = K. We will also use P to mean the number of taps in the synthesis
   filter, and N = KP to mean the size of the filter.

   The polarisations are computed completely independently.

   And, of course, xhat[n] is represented by the out array.
 */
{
    // First, set a generic variable for this thread
    int idx = blockDim.x*blockIdx.x + threadIdx.x;

    // The polarisation for this thread is
    int pol = idx % npol;

    // and the time index of the output (i.e. the index for xhat) is
    int n = idx / npol;

    // Other constants we'll need are:
    int K = blockDim.x / npol;  // Total number of channels (should be 128)
    int M = K;
    int P = ntaps;
    int F = P*K;

    // Because we must have 0 <= n-mM < F, the smallest allowed value of m
    // is:
    int m0 = (n - F)/M + 1;

    // Initialise the output sample to zero
    float out_real = 0.0;
    float out_imag = 0.0;

    // Perform the double sum
    int m, k, f, tw, ft, i;
    for (m = m0; m < m0 + P; m++)
    {
        // With m now known, we can get the index for the filter
        f = n - m*M;

        //printf("n=%d, m=%d, f=%d\n", n, m, f);
        for (k = 0; k < K; k++)
        {
            // The index for the twiddle factor is
            tw = ((k+K/2)*n) % K;
            // (the extra K/2 identifies the middle channel as the DC bin)

            // The index into the ft (= filter/twiddle) array is
            ft = F*tw + f;

            // The "in" index (see cu_invert_pfb() for how the in[] arrays
            // were packed)
            // The fine channel time index, m, must be adjusted to ensure that
            // n=0 corresponds to the first full filter's worth of input samples
            i = npol*K*(m+P) + npol*k + pol;

            // Complex multiplication
            out_real += in_real[i] * ft_real[ft] -
                        in_imag[i] * ft_imag[ft];
            out_imag += in_real[i] * ft_imag[ft] +
                        in_imag[i] * ft_real[ft];
        }
    }

    // out[] includes both polarisations, at adjacent indices
    out[2*idx]   = out_real / K;
    out[2*idx+1] = out_imag / K;

    __syncthreads();
}

void cu_invert_pfb( hipDoubleComplex ****detected_beam, int file_no,
                        int npointing, int nsamples, int nchan, int npol,
                        int sizeof_buffer,
                        struct gpu_ipfb_arrays *g, float *data_buffer_vdif )
/* "Invert the PFB" by applying a resynthesis filter, using GPU
 * acceleration.
 *
 * This function expects "detected_beam" to be structured as follows:
 *
 *   detected_beam[2*nsamples][nchan][npol]
 *
 * Although detected_samples potentially contains 2 seconds' worth of data,
 * this function only inverts 1 second. The appropriate second is worked out
 * using file_no: if it is even, the first half of detected_beam is used,
 * if odd, the second half.
 *
 * The output of the inversion is packed back into data_buffer_vdif, a 1D
 * array whose ordering is as follows:
 *
 *   time, pol, complexity
 *
 * This ordering is suited for immediate output to the VDIF format.
 *
 * It is assumed that the inverse filter coefficients have already been loaded
 * to the GPU.
 */
{
    // Setup input values:
    // The starting sample index is "ntaps" places from the end of the second
    // half of detected_beam if the file number is even, and "ntaps" places
    // from the end of the first half of detected_beam if the file number is
    // odd.
    
    int start_s = (file_no % 2 == 0 ? 2*nsamples - g->ntaps : nsamples - g->ntaps);

    int p, s_in, s, ch, pol, i;
    for (p = 0; p < npointing; p++)
    for (s_in = 0; s_in < nsamples + g->ntaps; s_in++)
    {
        s = (start_s + s_in) % (2*nsamples);
        for (ch = 0; ch < nchan; ch++)
        {
            for (pol = 0; pol < npol; pol++)
            {
                // Calculate the index for in_real and in_imag;
                i = p    * npol * nchan * (nsamples + g->ntaps) +
                    s_in * npol * nchan +
                    ch   * npol +
                    pol;
                // Copy the data across - taking care of the file_no = 0 case
                // The s_in%(npol*nchan*nsamples) does this for each pointing
                if (file_no == 0 && (s_in%(npol*nchan*nsamples)) < g->ntaps)
                {
                    g->in_real[i] = 0.0;
                    g->in_imag[i] = 0.0;
                }
                else
                {
                    g->in_real[i] = hipCreal( detected_beam[p][s][ch][pol] );
                    g->in_imag[i] = hipCimag( detected_beam[p][s][ch][pol] );
                }
            }
        }
    }
    
    // Copy the data to the device
    gpuErrchk(hipMemcpy( g->d_in_real, g->in_real, g->in_size, hipMemcpyHostToDevice ));
    gpuErrchk(hipMemcpy( g->d_in_imag, g->in_imag, g->in_size, hipMemcpyHostToDevice ));
    
    // Call the kernel
    if (npointing > 1)
    {
        fprintf( stderr, "error: PFB inversion currently only supports a single pointing\n" );
        exit(EXIT_FAILURE);
    }
    ipfb_kernel<<<nsamples, nchan*npol>>>( g->d_in_real, g->d_in_imag,
                                             g->d_ft_real, g->d_ft_imag,
                                             g->ntaps, npol, g->d_out );
    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();

    // Copy the result back into host memory
    gpuErrchk(hipMemcpy( data_buffer_vdif, g->d_out, g->out_size, hipMemcpyDeviceToHost ));
}


void cu_load_ipfb_filter( pfb_filter *filter, struct gpu_ipfb_arrays *g )
/* This function loads the inverse filter coefficients and the twiddle factors
   into GPU memory. If they were loaded separately (as floats), then the
   multiplication of the filter coefficients and the twiddle factors will be
   less precise than if a single array containing every combination of floats
   and twiddle factors is calculated in doubles, and then demoted to floats.
   Hence, this pre-calculation is done in this function before hipMemcpy is
   called.

   The result is 2x 1D arrays loaded onto the GPU (one for real, one for imag)
   where the ith element is equal to

   result[i] = f[n] * exp(2πjk/K),
   n = i % N  (N is the filter size, "fil_size")
   k = i / N
   and K is the number of channels (nchan).

*/
{
    int ch, f, i;

    // Setup filter values:
    hipDoubleComplex ft; // pre-calculated filter coeffs times twiddle factor
    hipDoubleComplex cf; // temp variable for complex version of filter coeffs
    for (f = 0; f < filter->size; f++)
    {
        cf = make_hipDoubleComplex( filter->coeffs[f], 0.0 );
        for (ch = 0; ch < filter->nchans; ch++)
        {
            i = filter->size*ch + f;
            ft = hipCmul( filter->twiddles[ch], cf );
            g->ft_real[i] = hipCreal( ft );
            g->ft_imag[i] = hipCimag( ft );
        }
    }

    gpuErrchk(hipMemcpy( g->d_ft_real, g->ft_real, g->ft_size, hipMemcpyHostToDevice ));
    gpuErrchk(hipMemcpy( g->d_ft_imag, g->ft_imag, g->ft_size, hipMemcpyHostToDevice ));
}


void malloc_ipfb( struct gpu_ipfb_arrays *g, pfb_filter *filter, int nsamples,
        int npol, int npointing )
{
    // Some shorthand variables:
    int ntaps = filter->ntaps;
    int nchan = filter->nchans;
    int fil_size = filter->size;

    // Flatten the input array (detected_array) for GPU.
    // We only need one second's worth, plus 12 time samples tacked onto the
    // beginning (from the previous second)

    g->ntaps     = ntaps;
    g->in_size   = npointing * ((nsamples + ntaps) * nchan * npol) * sizeof(float);
    g->ft_size   = fil_size * nchan * sizeof(float);
    g->out_size  = npointing * nsamples * filter->nchans * npol * 2 * sizeof(float);

    // Allocate memory on the device
    gpuErrchk(hipMalloc( (void **)&g->d_in_real, g->in_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_in_imag, g->in_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_ft_real, g->ft_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_ft_imag, g->ft_size ));

    gpuErrchk(hipMalloc( (void **)&g->d_out, g->out_size ));

    // Allocate memory for host copies of the same
    g->in_real = (float *)malloc( g->in_size );
    g->in_imag = (float *)malloc( g->in_size );
    g->ft_real = (float *)malloc( g->ft_size );
    g->ft_imag = (float *)malloc( g->ft_size );

}


void free_ipfb( struct gpu_ipfb_arrays *g )
{
    // Free memory on host and device
    free( g->in_real );
    free( g->in_imag );
    free( g->ft_real );
    free( g->ft_imag );
    hipFree( g->d_in_real );
    hipFree( g->d_in_imag );
    hipFree( g->d_ft_real );
    hipFree( g->d_ft_imag );
    hipFree( g->d_out );
}
